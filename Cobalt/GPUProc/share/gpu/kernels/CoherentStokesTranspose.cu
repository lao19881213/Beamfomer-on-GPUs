#include "hip/hip_runtime.h"
//# CoherentStokesTranspose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: CoherentStokesTranspose.cu 27262 2013-11-06 13:11:17Z klijn $

/*!
 * Performs data transposition from the output of the beamformer kernel to
 * a data order suitable for an inverse FFT.
 * Parallelisation is performed over the TABs and number of samples (time).
 *
 *
 * \param[out] OutputDataType      4D output array of samples. For each TAB and pol, a spectrum per time step of complex floats.
 * \param[in]  InputDataType       3D input array of samples (last dim (pol) is implicit). For each channel, the TABs per time step of two complex floats.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_SAMPLES_PER_CHANNEL  | >= 1                    | number of input samples per channel
 * NR_CHANNELS             | multiple of 16 and > 0  | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create, multiple 16 is optimal
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2
 *
 * Execution configuration:
 * - LocalWorkSize = 2 dimensional; (16, 16, 1) is in use.
 * - GlobalWorkSize = 3 dimensional:
 *   + inner dim (x): nr (( params.nrTABs + 16 - 1) / 16) * 16 
 *   + middle dim (y): nr samples ( /16)
 *   + outer dim (z): number of channels (/1)
 */
#include "gpu_math.cuh"

#if !(NR_SAMPLES_PER_CHANNEL >= 1)
#error Precondition violated: NR_SAMPLES_PER_CHANNEL >= 1
#endif

#if !(NR_TABS >= 1)
#error Precondition violated: NR_TABS >= 1
#endif

#if !(NR_CHANNELS >= 16)
#error Precondition violated: NR_CHANNELS >= 16
#endif

typedef fcomplex (*OutputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 

typedef float4 fcomplex2;
// Allows for better memory access
typedef fcomplex2 (*InputDataType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS]; // [NR_POLARIZATIONS];


extern "C"
__global__ void coherentStokesTranspose(void *OutputDataPtr,
                          const void *InputDataPtr)
{ 
  OutputDataType outputData = (OutputDataType) OutputDataPtr;
  InputDataType inputData = (InputDataType) InputDataPtr;
  
  unsigned tab      = blockIdx.x * blockDim.x + threadIdx.x;  
  unsigned channel  =  blockIdx.y * blockDim.y + threadIdx.y;
  unsigned sample       = blockIdx.z * blockDim.z ;  

  // Use shared memory for the transpose
  __shared__ fcomplex2 tmp[16][16 + 1];  // plus one to prevent bank conflicts in shared memory

  // get the data if the current tab exists
  if ( tab < NR_TABS) 
    tmp[threadIdx.y][threadIdx.x] = (*inputData) [channel][sample][tab];

  __syncthreads();  // ensures all writes are done
  
  // Reassign the tab and sample to allow the threadIdx.x to write in the highest dimension
  tab           = blockIdx.x* blockDim.x + threadIdx.y;
  channel        = blockIdx.y * blockDim.y + threadIdx.x;

  // Do the write to global mem if the current tab exists
  if ( tab < NR_TABS) 
  {
    (*outputData)[tab][0][sample][channel] = make_float2(tmp[threadIdx.x][threadIdx.y].x,
      tmp[threadIdx.x][threadIdx.y].y) ;
    (*outputData)[tab][1][sample][channel] = make_float2(tmp[threadIdx.x][threadIdx.y].z,
      tmp[threadIdx.x][threadIdx.y].w) ;
  }

  __syncthreads();  // ensures all writes are done
}
